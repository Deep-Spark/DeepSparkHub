#include "hip/hip_runtime.h"
#include <c10/cuda/CUDACachingAllocator.h>
#include <c10/core/ScalarType.h>
#include <thrust/pair.h>
#include <thrust/execution_policy.h>
#include "gn_kernel.h"
#include "Welford.h"
#include "vecs.h"
#define MAX_THREADS_PER_BLOCK 512 // 512 slightly faster (~3%) than 1024 because of higher theoretical occupancy -> higher mem throughput
#define MAX(a, b) (a > b) ? a : b
#define MIN(a, b) (a < b) ? a : b

#define DEBUG_ENABLED 0
#if DEBUG_ENABLED
#define DEBUG(format, args...) fprintf(stderr, format, args)
#else
#define DEBUG(format, args...) ((void)0)
#endif
#define ELEM_DEBUG 0
#define INT int // torch uses int64_t but this came at a pretty big hit to performance and the input sizes that I frequently use (resolutions no bigger than 1024x1024) have a number of pixels smaller than the int max value

template <typename T>
struct acc_type { using type = float; };
template <>
struct acc_type<double> { using type = double; };

typedef struct block_params {
  int t; // threads per block
  int d; // dimensionality (number of rows of data that each threadblock proceesses in parallel)
  int f; // factor (number of different threadblocks needed to represent one row of data) 
} block_params_t;

inline block_params_t calc_block_params(const int ideal_num_threads, const int threads_per_row, int f_divides = -1, const int tpb_divides = -1) {
  /*
  ideal_num_threads: absolute upper limit of threads that a block should have (e.g. a kernel that operates on only 30 elements should have a max TPB of 30 (ideal_num_threads=30))
  threads_per_row: determines the user-specified upper limit on the size of blockDim.x
    - meant to be set to the size of the last dimension, e.g. a kernel operating on tensor sized (N, R, C) would have threads_per_row=C
  f_divides: optional parameter if user needs to explicitly specify a stricter requirement on the divisibility of the number of threads per block
    - e.g. fwd with C = 2560, G = 32, TPB = 480 wouldn't work since that means 32 groups are split over f=5 blocks (5.333 groups per block)
    - e.g. fwd with C = 2560, G = 32, TPB = 320 would work since that means 32 groups are split over f=8 blocks (4 groups per block), you could say that f divides 32 (f_divides=32)
  tpb_divides: optional parameter if user needs to explicitly specify that the returned threads per block needs to divide another value (e.g. a kernel where bounds checking isn't implemented)
    - e.g. fwd with H, W, C = 5, 5, 32; TPB = 512 wouldn't work since that means you use 1.5625 blocks to represent H*W*C (800) elements
    - e.g. fwd with H, W, C = 5, 5, 32; TPB = 160 would work since that means you use 5 blocks to represent H*W*C (800) elements, you could say that TPB (160) divides 800 (tpb_divides=800)
  */
  int TPB, d = 1, f = 1;
  f_divides = f_divides == -1 ? threads_per_row : f_divides;
  TPB = MIN(MAX_THREADS_PER_BLOCK, ideal_num_threads);
  if (threads_per_row < TPB) {
    d = TPB / threads_per_row;
    if (tpb_divides != -1) // could be put as another condition in the while loop but it hurts readability
      while (tpb_divides % (threads_per_row * d) != 0) // d = 1 guaranteed to break this condition
        --d;
  }
  else
    while (f_divides % f != 0 || threads_per_row / f > MAX_THREADS_PER_BLOCK)
      ++f;
  TPB = threads_per_row * d / f;
  return {TPB, d, f};
}

template <typename T> __device__ T inline identity(T x) {
  return x;
}
template <typename T> __device__ T inline identity_d(T /*x*/) {
  return 1;
}

template <typename T> __device__ T inline relu(T x) {
  return x > 0 ? x : static_cast<T>(0);
}
template <typename T> __device__ T inline relu_d(T x) {
  return x > 0 ? static_cast<T>(1) : static_cast<T>(0);
}

template <typename T> __device__ T inline silu(T x) {
  return x / (1 + exp(-x));
}
template <typename T> __device__ T inline silu_d(T x) {
  const T s = 1 / (1 + exp(-x));
  return s * (1 + x * (1 - s));
}

template <typename T> __device__ T inline gelu(T x) {
  constexpr float kAlpha = M_SQRT1_2;
  return x * T(0.5) * (T(1) + erf(x * kAlpha));
}
template <typename T> __device__ T inline gelu_d(T x) {
  constexpr float kBeta = M_2_SQRTPI * M_SQRT1_2 * 0.5;
  constexpr float kAlpha = M_SQRT1_2;
  const T cdf = T(0.5) * (T(1) + erf(x * kAlpha));
  const T pdf = exp(T(-0.5) * x * x) * kBeta;
  return cdf + x * pdf;
}

template <typename T> __device__ T inline gelu_tanh(T x) {
  constexpr float kBeta = M_SQRT2 * M_2_SQRTPI * 0.5;
  constexpr float kKappa = 0.044715;
  auto inner = kBeta * (x + kKappa * x * x * x);
  return T(0.5) * x * (T(1) + tanh(inner));
}
template <typename T> __device__ T inline gelu_tanh_d(T x) {
  constexpr float kBeta = M_SQRT2 * M_2_SQRTPI * 0.5;
  constexpr float kKappa = 0.044715;
  auto x_sq = x * x;
  auto x_cube = x_sq * x;
  auto inner = kBeta * (x + kKappa * x_cube);
  auto tanh_inner = tanh(inner);

  auto left = T(0.5) * x;
  auto right = T(1) + tanh_inner;

  auto left_derivative = T(0.5) * right;

  auto tanh_derivative = T(1) - tanh_inner * tanh_inner;
  auto inner_derivative = kBeta * (T(1) + T(3) * kKappa * x_sq);
  auto right_derivative = left * tanh_derivative * inner_derivative;

  return left_derivative + right_derivative;
}

//////////////////////////////////////////////////
// forward kernels
//////////////////////////////////////////////////

template <typename T>
__global__ void
compute_stats_pt1(
    const T* X,
    const int H,
    const int W,
    const int C,
    const int G,
    WelfordData<typename acc_type<T>::type, INT> *welford_data
  ) {
  /*
  Computes means and rstds of X on the W (width) dimension.
  grid: (x=N, y=H, z=f); block: (x=TPB/d, y=d)
  - TPB = Cd/f
  if TPB < C (f > 1, d=1)
    C = f*TPB
    X shape: (N, H, W, C) -view-> (N, H, W, 1, f, TPB); X stride: (HWC, WC, C, C, TPB, 1)
    dram reduction (per block): (W, 1, TPB) -reduce-> (1, TPB)
  else (block.x=C, block.y=d)
    TPB = Cd
    X shape: (N, H, W, C) -view-> (N, H, W/d, d, 1, C); X stride: (HWC, WC, dC, C, C, 1)
    dram reduction (per block): (W/d, d, C) -reduce-> (d, C)
  shmem reduction (per block): (TPB,) -view-> (d, G/f, D) -permute-> (d, D, G/f) -reduce-> G/f
  output buffer: (N, f, G/f, H)
  */
  using T_ACC = typename acc_type<T>::type;
  using WelfordType = WelfordData<T_ACC, INT>;
  using WelfordOp = WelfordOps<T_ACC, T_ACC, INT, thrust::pair<T_ACC, T_ACC>>;
  const int TPB = blockDim.y * blockDim.x;
  const int d = blockDim.y;

  WelfordOp welford_op = {/*correction=*/0, /*take_sqrt=*/false};
  WelfordType val(0, 0, 0, 0);

  const int w = ceil((float)W / d);
  int i;
#pragma unroll
  for (i = 0; i < w - 1; ++i) {
    int reduce_idx = 0;
    reduce_idx += blockIdx.x * H * W * C;
    reduce_idx += blockIdx.y * W * C;
    reduce_idx += i * d * C;
    reduce_idx += threadIdx.y * C;
    reduce_idx += blockIdx.z * TPB;
    reduce_idx += threadIdx.x;
    T x = X[reduce_idx];
    val = welford_op.reduce(val, static_cast<T_ACC>(x));
  }
  if ((int)(i * d + threadIdx.y) < W) // last iteration to deal with inputs with weird width sizes
    val = welford_op.reduce(val, static_cast<T_ACC>(X[blockIdx.x * H * W * C + blockIdx.y * W * C + i * d * C + threadIdx.y * C + blockIdx.z * TPB + threadIdx.x]));

  // shmem reduction
  const int D = C / G;
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;
  const int f = gridDim.z;
  const int gf = G / f;
  const int d_idx = threadIdx.y;
  const int gf_idx = threadIdx.x / D;
  const int D_idx = threadIdx.x % D;

  __shared__ typename std::aligned_storage<sizeof(WelfordType), alignof(WelfordType)>::type vals_reduced_arr[MAX_THREADS_PER_BLOCK];
  WelfordType *vals_reduced = reinterpret_cast<WelfordType*>(vals_reduced_arr);

  int idx = 0;
  idx += d_idx * D * gf;
  idx += D_idx * gf;
  idx += gf_idx;
  vals_reduced[idx] = val;
  __syncthreads();

  int reduce_n = TPB / gf; // number of inputs that gets reduced to a single output
#pragma unroll
  for (int stride = TPB / 2; stride >= gf && reduce_n % 2 == 0 && stride % gf == 0; stride >>= 1, reduce_n >>= 1) {
    if (tid < stride)
      vals_reduced[tid] = welford_op.combine(vals_reduced[tid], vals_reduced[tid + stride]);
    __syncthreads();
  }

  // put reduced outputs into return buffers
  if (tid < gf) {
#pragma unroll
    for (int i = 1; i < reduce_n; ++i)
      vals_reduced[tid] = welford_op.combine(vals_reduced[tid], vals_reduced[tid + i * gf]);

    int out_idx = 0;
    out_idx += blockIdx.x * G * H;
    out_idx += blockIdx.z * gf * H;
    out_idx += tid * H;
    out_idx += blockIdx.y;
    welford_data[out_idx] = vals_reduced[tid];
  }
}

template <typename T>
__global__ void
compute_stats_pt2(
    WelfordData<typename acc_type<T>::type, INT> *welford_data,
    const int H,
    const int G,
    const T eps,
    T* means,
    T* rstds
  ) {
  using T_ACC = typename acc_type<T>::type;
  using WelfordType = WelfordData<T_ACC, INT>;
  using WelfordOp = WelfordOps<T_ACC, T_ACC, INT, thrust::pair<T_ACC, T_ACC>>;
  /*
  Computes means and rstds of X on the H (height) dimension.
  grid: (x=N, y=G); block: (x=H/f)
  - TPB = Gd/f
  welford_data shape: (N, G, H) -view-> (N, G, f, H/f); X stride: (GH, H, H/f, 1)
  dram reduction (per block): (f, H/f) -reduce-> (H/f,)
  shmem reduction (per block): (H/f) -reduce-> (1,)
  output buffer: (N, G)
  */

  WelfordOp welford_op = {/*correction=*/0, /*take_sqrt=*/false};
  WelfordType val(0, 0, 0, 0);
  const int TPB = blockDim.y * blockDim.x;

  const int f = H / TPB;
  for (int i = 0 ; i < f; ++i) {
    int idx = 0;
    idx += blockIdx.x * G * H;
    idx += blockIdx.y * H;
    idx += i * H / f;
    idx += threadIdx.x;
    val = welford_op.combine(val, welford_data[idx]);
  }

  // shmem reduction
  __shared__ typename std::aligned_storage<sizeof(WelfordType), alignof(WelfordType)>::type vals_reduced_arr[MAX_THREADS_PER_BLOCK];
  WelfordType *vals_reduced = reinterpret_cast<WelfordType*>(vals_reduced_arr);

  const int tid = threadIdx.x;
  vals_reduced[tid] = val;
  __syncthreads();

  int reduce_n = TPB; // number of inputs that gets reduced to a single output

#pragma unroll
  for (int stride = TPB / 2; stride >= 1 && reduce_n % 2 == 0; stride >>= 1, reduce_n >>= 1) {
    if (tid < stride)
      vals_reduced[tid] = welford_op.combine(vals_reduced[tid], vals_reduced[tid + stride]);
    __syncthreads();
  }

  // put reduced outputs into return buffers
  if (tid == 0) {
#pragma unroll
    for (int i = 1; i < reduce_n; ++i)
      vals_reduced[tid] = welford_op.combine(vals_reduced[tid], vals_reduced[tid + i]);

    T_ACC mean, var;
    thrust::tie(var, mean) = welford_op.project(vals_reduced[tid]);
    int out_idx = 0;
    out_idx += blockIdx.x * G;
    out_idx += blockIdx.y;
    means[out_idx] = mean;
    rstds[out_idx] = rsqrt(var + static_cast<T_ACC>(eps));
  }
}

template <typename T, int LOOP_I, int vec_elems, int64_t act_fn_option>
__global__ void
scale_shift(
    const T* X_data,
    const T* mean_data,
    const T* rstd_data,
    const T* weight_data,
    const T* bias_data,
    const int N,
    const int C,
    const int G,
    T* y
    ) {
  /*
  Performs elementwise op (X - mean) * rstd * weight + bias. Vectorized for speed.
  LOOP_I: number of elements that each thread processes.
  vec_elems: number of elements stored for each vector.
  grid: (x=NHWC / (TPB * LOOP_I * f), y=f), block: (x=TPB)
  - HWC % (TPB * LOOP_I * f) = 0
  - TPB * f % C = 0
  X shape: (N, H, W, C) -view-> (NHWC / (TPB * LOOP_I * f), LOOP_I, f, TPB); X.stride: (LOOP_I * f * TPB, f * TPB, TPB, 1)
  */
  using T_ACC = typename acc_type<T>::type;
  using V = float_vec<T, vec_elems>;
  const int f = gridDim.y;
  const int TPB = blockDim.x;

  const int n = (N * blockIdx.x) / gridDim.x;
  const int c = (blockIdx.y * blockDim.x + threadIdx.x) % (C / vec_elems);
  const int g = (G * c) / (C / vec_elems);
  const int ng = n * G + g;
  const V *X_vecs = reinterpret_cast<const V*>(X_data);
  const V *weight_vecs = reinterpret_cast<const V*>(weight_data);
  const V *bias_vecs = reinterpret_cast<const V*>(bias_data);
  V *y_vecs = reinterpret_cast<V*>(y);
  T mean = mean_data[ng];
  T rstd = rstd_data[ng];
  V weight_vec = weight_vecs[c];
  V bias_vec = bias_vecs[c];

  // compute fused weight/bias a,b such that (x - mean) * rstd * weight + bias = x * a + b
  V fused_weight, fused_bias;
  if constexpr (vec_elems == 1) {
    fused_weight = {rstd * weight_vec.x};
    fused_bias = {-mean * fused_weight.x + bias_vec.x};
  }
  else if constexpr (vec_elems == 2) {
    fused_weight = {
      rstd * weight_vec.x,
      rstd * weight_vec.y
    };
    fused_bias = {
      -mean * fused_weight.x + bias_vec.x,
      -mean * fused_weight.y + bias_vec.y
    };
  }
  else if constexpr (vec_elems == 4) {
    fused_weight = {
      rstd * weight_vec.x,
      rstd * weight_vec.y,
      rstd * weight_vec.z,
      rstd * weight_vec.w
    };
    fused_bias = {
      -mean * fused_weight.x + bias_vec.x,
      -mean * fused_weight.y + bias_vec.y,
      -mean * fused_weight.z + bias_vec.z,
      -mean * fused_weight.w + bias_vec.w
    };
  }

  T (*act_fn)(T);
  if constexpr (act_fn_option == 0)
    act_fn = identity;
  else if constexpr (act_fn_option == 1)
    act_fn = relu;
  else if constexpr (act_fn_option == 2)
    act_fn = silu;
  else if constexpr (act_fn_option == 3)
    act_fn = gelu;
  else if constexpr (act_fn_option == 4)
    act_fn = gelu_tanh;

#pragma unroll
  for (int i = 0; i < LOOP_I; ++i) {
    int idx = 0;
    idx += blockIdx.x * LOOP_I * f * TPB;
    idx += i * f * TPB;
    idx += blockIdx.y * TPB;
    idx += threadIdx.x;
    V X_vec = X_vecs[idx];
    
    if constexpr (vec_elems == 1)
      y_vecs[idx] = {act_fn(static_cast<T_ACC>(X_vec.x) * fused_weight.x + fused_bias.x)};
    else if constexpr (vec_elems == 2) {
      y_vecs[idx] = {
        act_fn(static_cast<T_ACC>(X_vec.x) * fused_weight.x + fused_bias.x),
        act_fn(static_cast<T_ACC>(X_vec.y) * fused_weight.y + fused_bias.y),
      };
    }
    else if constexpr (vec_elems == 4) {
      y_vecs[idx] = {
        act_fn(static_cast<T_ACC>(X_vec.x) * fused_weight.x + fused_bias.x),
        act_fn(static_cast<T_ACC>(X_vec.y) * fused_weight.y + fused_bias.y),
        act_fn(static_cast<T_ACC>(X_vec.z) * fused_weight.z + fused_bias.z),
        act_fn(static_cast<T_ACC>(X_vec.w) * fused_weight.w + fused_bias.w),
      };
    }
  }
}

template <typename T>
void run_gn_fwd_kernels(
    const T *X_data,
    const T *weight_data,
    const T *bias_data,
    const int N,
    const int H,
    const int W,
    const int C,
    const int G,
    T eps,
    const int64_t act_fn_option,
    T *Y_data,
    T *mean_data,
    T *rstd_data) {
  using T_ACC = typename acc_type<T>::type;
  using WelfordType = WelfordData<T_ACC, INT>;
  WelfordType *welford_data = (WelfordType*)c10::cuda::CUDACachingAllocator::raw_alloc(sizeof(WelfordType) * N * G * H);
  hipStream_t hip_stream = at::cuda::getCurrentCUDAStream();
  
  // compute means/rstds over width dimension
  {
    auto [TPB, d, f] = calc_block_params(W * C, C, G);
    DEBUG("starting compute_stats 1, N: %d, H: %d, W: %d, C: %d, G: %d, D: %d, TPB: %d, d: %d, f: %d, G/f: %d\n", N, H, W, C, G, (C / G), TPB, d, f, (G / f));
    compute_stats_pt1<<<dim3(N, H, f), dim3(TPB / d, d), 0, hip_stream>>>(
        X_data,
        H, W, C, G, 
        welford_data
    );
  }

  // compute means/rstds over height dimension
  {
    auto [TPB, d, f] = calc_block_params(H, H);
    DEBUG("starting compute_stats 2, N: %d, H: %d, W: %d, C: %d, G: %d, D: %d, TPB: %d, d: %d, f: %d, G/f: %d\n", N, H, W, C, G, (C / G), TPB, d, f, (G / f));
    compute_stats_pt2<<<dim3(N, G), H / f, 0, hip_stream>>>(
        welford_data,
        H, G, eps,
        mean_data, rstd_data
    );
  }

  // scale/shift X
  {
    const int D = C / G;
    int vec_elems;
    if (D % 4 == 0) vec_elems = 4;
    else if (D % 2 == 0) vec_elems = 2;
    else vec_elems = 1;
    auto [TPB, d, f] = calc_block_params(H * W * C / 8 / vec_elems, C);

    if (!ELEM_DEBUG && ((H * W * C) % (TPB * 8 * f * vec_elems) == 0)) {
      const int LOOP_I = 8;
      const int num_blocks = N * H * W * C / TPB / LOOP_I / f;
      DEBUG("scale shift starting (LOOP_I = 8), N: %d, H: %d, W: %d, C: %d, G: %d, D: %d, TPB: %d, f: %d, num blocks (before vectors): %d, vec_elems: %d\n", N, H, W, C, G, D, TPB, f, num_blocks, vec_elems);
      if (vec_elems == 4 && act_fn_option == 0) // i'm sorry
        scale_shift<T, LOOP_I, 4, 0><<<dim3(num_blocks / vec_elems, f), TPB, 0, hip_stream>>>(X_data, mean_data, rstd_data, weight_data, bias_data, N, C, G, Y_data);
      else if (vec_elems == 2 && act_fn_option == 0)
        scale_shift<T, LOOP_I, 2, 0><<<dim3(num_blocks / vec_elems, f), TPB, 0, hip_stream>>>(X_data, mean_data, rstd_data, weight_data, bias_data, N, C, G, Y_data);
      else if (vec_elems == 1 && act_fn_option == 0)
        scale_shift<T, LOOP_I, 1, 0><<<dim3(num_blocks / vec_elems, f), TPB, 0, hip_stream>>>(X_data, mean_data, rstd_data, weight_data, bias_data, N, C, G, Y_data);
      else if (vec_elems == 4 && act_fn_option == 1)
        scale_shift<T, LOOP_I, 4, 1><<<dim3(num_blocks / vec_elems, f), TPB, 0, hip_stream>>>(X_data, mean_data, rstd_data, weight_data, bias_data, N, C, G, Y_data);
      else if (vec_elems == 2 && act_fn_option == 1)
        scale_shift<T, LOOP_I, 2, 1><<<dim3(num_blocks / vec_elems, f), TPB, 0, hip_stream>>>(X_data, mean_data, rstd_data, weight_data, bias_data, N, C, G, Y_data);
      else if (vec_elems == 1 && act_fn_option == 1)
        scale_shift<T, LOOP_I, 1, 1><<<dim3(num_blocks / vec_elems, f), TPB, 0, hip_stream>>>(X_data, mean_data, rstd_data, weight_data, bias_data, N, C, G, Y_data);
      else if (vec_elems == 4 && act_fn_option == 2)
        scale_shift<T, LOOP_I, 4, 2><<<dim3(num_blocks / vec_elems, f), TPB, 0, hip_stream>>>(X_data, mean_data, rstd_data, weight_data, bias_data, N, C, G, Y_data);
      else if (vec_elems == 2 && act_fn_option == 2)
        scale_shift<T, LOOP_I, 2, 2><<<dim3(num_blocks / vec_elems, f), TPB, 0, hip_stream>>>(X_data, mean_data, rstd_data, weight_data, bias_data, N, C, G, Y_data);
      else if (vec_elems == 1 && act_fn_option == 2)
        scale_shift<T, LOOP_I, 1, 2><<<dim3(num_blocks / vec_elems, f), TPB, 0, hip_stream>>>(X_data, mean_data, rstd_data, weight_data, bias_data, N, C, G, Y_data);
      else if (vec_elems == 4 && act_fn_option == 3)
        scale_shift<T, LOOP_I, 4, 3><<<dim3(num_blocks / vec_elems, f), TPB, 0, hip_stream>>>(X_data, mean_data, rstd_data, weight_data, bias_data, N, C, G, Y_data);
      else if (vec_elems == 2 && act_fn_option == 3)
        scale_shift<T, LOOP_I, 2, 3><<<dim3(num_blocks / vec_elems, f), TPB, 0, hip_stream>>>(X_data, mean_data, rstd_data, weight_data, bias_data, N, C, G, Y_data);
      else if (vec_elems == 1 && act_fn_option == 3)
        scale_shift<T, LOOP_I, 1, 3><<<dim3(num_blocks / vec_elems, f), TPB, 0, hip_stream>>>(X_data, mean_data, rstd_data, weight_data, bias_data, N, C, G, Y_data);
      else if (vec_elems == 4 && act_fn_option == 4)
        scale_shift<T, LOOP_I, 4, 4><<<dim3(num_blocks / vec_elems, f), TPB, 0, hip_stream>>>(X_data, mean_data, rstd_data, weight_data, bias_data, N, C, G, Y_data);
      else if (vec_elems == 2 && act_fn_option == 4)
        scale_shift<T, LOOP_I, 2, 4><<<dim3(num_blocks / vec_elems, f), TPB, 0, hip_stream>>>(X_data, mean_data, rstd_data, weight_data, bias_data, N, C, G, Y_data);
      else if (vec_elems == 1 && act_fn_option == 4)
        scale_shift<T, LOOP_I, 1, 4><<<dim3(num_blocks / vec_elems, f), TPB, 0, hip_stream>>>(X_data, mean_data, rstd_data, weight_data, bias_data, N, C, G, Y_data);
    }
    else {// relatively slow fallback
      const int num_blocks = N * H * W;
      DEBUG("SLOW FALLBACK, scale shift kernel starting, N: %d, H: %d, W: %d, C: %d, G: %d, D: %d, TPB: %d, f: %d, num blocks (before vectors): %d, vec_elems: %d\n", N, H, W, C, G, D, C/f, f, num_blocks, vec_elems);
      if (act_fn_option == 0)
        scale_shift<T, 1, 1, 0><<<dim3(num_blocks, f), C / f, 0, hip_stream>>>(X_data, mean_data, rstd_data, weight_data, bias_data, N, C, G, Y_data);
      if (act_fn_option == 1)
        scale_shift<T, 1, 1, 1><<<dim3(num_blocks, f), C / f, 0, hip_stream>>>(X_data, mean_data, rstd_data, weight_data, bias_data, N, C, G, Y_data);
      if (act_fn_option == 2)
        scale_shift<T, 1, 1, 2><<<dim3(num_blocks, f), C / f, 0, hip_stream>>>(X_data, mean_data, rstd_data, weight_data, bias_data, N, C, G, Y_data);
      if (act_fn_option == 3)
        scale_shift<T, 1, 1, 3><<<dim3(num_blocks, f), C / f, 0, hip_stream>>>(X_data, mean_data, rstd_data, weight_data, bias_data, N, C, G, Y_data);
      if (act_fn_option == 4)
        scale_shift<T, 1, 1, 4><<<dim3(num_blocks, f), C / f, 0, hip_stream>>>(X_data, mean_data, rstd_data, weight_data, bias_data, N, C, G, Y_data);
    }
  }

  c10::cuda::CUDACachingAllocator::raw_delete(welford_data);
}

template void run_gn_fwd_kernels<float>(const float *X_data, const float *weight_data, const float *bias_data, const int N, const int h, const int W, const int C, const int G, float eps, const int64_t act_fn_option, float *Y_data, float *mean_data, float *rstd_data);
template void run_gn_fwd_kernels<double>(const double *X_data, const double *weight_data, const double *bias_data, const int N, const int h, const int W, const int C, const int G, double eps, const int64_t act_fn_option, double *Y_data, double *mean_data, double *rstd_data);
template void run_gn_fwd_kernels<c10::Half>(const c10::Half *X_data, const c10::Half *weight_data, const c10::Half *bias_data, const int N, const int h, const int W, const int C, const int G, c10::Half eps, const int64_t act_fn_option, c10::Half *Y_data, c10::Half *mean_data, c10::Half *rstd_data);
template void run_gn_fwd_kernels<c10::BFloat16>(const c10::BFloat16 *X_data, const c10::BFloat16 *weight_data, const c10::BFloat16 *bias_data, const int N, const int h, const int W, const int C, const int G, c10::BFloat16 eps, const int64_t act_fn_option, c10::BFloat16 *Y_data, c10::BFloat16 *mean_data, c10::BFloat16 *rstd_data);

//////////////////////////////////////////////////
// backward kernels
//////////////////////////////////////////////////

template <typename T>
__device__ void
sum_reduce(
    T vals_reduced,
    const int start_stride,
    const int end_stride
  ) {
  // Sums a shared buffer (vals_reduced) with shape (2 * start_stride / end_stride, end_stride) into (end_stride,).
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;
  int reduce_n = 2 * start_stride / end_stride;

#pragma unroll
  for (int stride = start_stride; stride >= end_stride && reduce_n % 2 == 0 && stride % end_stride == 0; stride >>= 1, reduce_n >>= 1) {
    if (tid < stride)
      vals_reduced[tid] += vals_reduced[tid + stride];
    __syncthreads();
  }

  if (tid < end_stride)
#pragma unroll
    for (int i = 1; i < reduce_n; ++i)
      vals_reduced[tid] += vals_reduced[tid + i * end_stride];
  __syncthreads();
}

template <typename T, int64_t act_fn_option>
__global__ void
width_reduce(
      const T* dy_data,
      const T* X_data,
      const T* mean_data,
      const T* rstd_data,
      const T* weight_data,
      const T* bias_data,
      const int H,
      const int W,
      const int C,
      const int G,
      typename acc_type<T>::type *xdy_dy_sum_data) {
  /*
  Loops over W (width) dimension, loading and summing dy, X, and the activation derivative of Y. Outputs stored in xdy_dy_sum_data. Spatial dimension H is processed in a separate kernel.
  grid: (x=N, y=H, z=f); blockdim: (x=TPB/d, y=d)
    TPB = Cd/f
  if TPB < C (f > 1, d=1)
    C = f*TPB
    X shape: (N, H, W, C) -view-> (N, H, W, 1, f, TPB); X stride: (HWC, WC, C, C, TPB, 1)
    dram reduction (per block): (W, 1, TPB) -reduce-> (TPB,)
  else (block.x=C, block.y=d)
    TPB = Cd
    X shape: (N, H, W, C) -view-> (N, H, W/d, d, 1, C); X stride: (HWC, WC, dC, C, C, 1)
    dram reduction (per block): (W/d, d, C) -reduce-> (d, C)
  shmem reduction (per block): (TPB, 2) -> (d, C/f, 2) -reduce-> (C/f, 2) (the 2 comes from storing both xdy_sum and dy_sum in the same buffer)
  output buffer: (N, f, C/f, H, 2) -view-> (N, C, H, 2)
    xdy_dy_sum_data[:, :, :, 0] = x * dy * activation_derivative((x-mean)*rstd*weight+bias)
    xdy_dy_sum_data[:, :, :, 1] = dy * activation_derivative((x-mean)*rstd*weight+bias)
   */
  using T_ACC = typename acc_type<T>::type;

  const int TPB = blockDim.y * blockDim.x;
  const int d = blockDim.y;
  T_ACC xdy_sum = 0;
  T_ACC dy_sum = 0;

  const int n = blockIdx.x;
  int c = blockIdx.z * blockDim.x + threadIdx.x;
  int g = G * c / C;
  const int ng = n * G + g;
  T_ACC fused_scale = rstd_data[ng] * weight_data[c];
  T_ACC fused_bias = -mean_data[ng] * fused_scale + bias_data[c];

  T (*act_d_fn)(T x);
  if constexpr (act_fn_option == 0)
    act_d_fn = identity_d;
  else if constexpr (act_fn_option == 1)
    act_d_fn = relu_d;
  else if constexpr (act_fn_option == 2)
    act_d_fn = silu_d;
  else if constexpr (act_fn_option == 3)
    act_d_fn = gelu_d;
  else if constexpr (act_fn_option == 4)
    act_d_fn = gelu_tanh_d;

  const int w = ceil((float)W / d);
  int i;
#pragma unroll
  for (i = 0; i < w - 1; ++i) {
    int reduce_idx = 0;
    reduce_idx += blockIdx.x * H * W * C;
    reduce_idx += blockIdx.y * W * C;
    reduce_idx += i * d * C;
    reduce_idx += threadIdx.y * C;
    reduce_idx += blockIdx.z * TPB;
    reduce_idx += threadIdx.x;
    T_ACC dy_elem = static_cast<T_ACC>(dy_data[reduce_idx]);
    T_ACC X_elem = static_cast<T_ACC>(X_data[reduce_idx]);
    T_ACC X_norm = X_elem * fused_scale + fused_bias;
    T_ACC d_act = act_d_fn(X_norm);
    xdy_sum += dy_elem * X_elem * d_act;
    dy_sum += dy_elem * d_act;
  }
  if ((int)(i * d + threadIdx.y) < W) { // last iteration to deal with inputs with weird width sizes
    int reduce_idx = blockIdx.x * H * W * C + blockIdx.y * W * C + i * d * C + threadIdx.y * C + blockIdx.z * TPB + threadIdx.x;
    T_ACC dy_elem = static_cast<T_ACC>(dy_data[reduce_idx]);
    T_ACC X_elem = static_cast<T_ACC>(X_data[reduce_idx]);
    T_ACC X_norm = X_elem * fused_scale + fused_bias;
    T_ACC d_act = act_d_fn(X_norm);
    xdy_sum += dy_elem * X_elem * d_act;
    dy_sum += dy_elem * d_act;
  }

  // shmem reduction
  extern __shared__ char vals_reduced_uncasted[]; // size 2*TPB, TPB for sum1, TPB for sum2
  T_ACC *vals_reduced = reinterpret_cast<T_ACC*>(vals_reduced_uncasted);

  const int tid = threadIdx.y * blockDim.x + threadIdx.x;
  vals_reduced[2 * tid] = xdy_sum;
  vals_reduced[2 * tid + 1] = dy_sum;
  __syncthreads();
  sum_reduce(vals_reduced, TPB, 2 * C);

  // put reduced outputs into return buffers
  if (tid < C) {
    int out_idx = 0;
    out_idx += blockIdx.x * C * H;
    out_idx += (blockIdx.z * TPB + tid) * H;
    out_idx += blockIdx.y;

    xdy_dy_sum_data[2 * out_idx] = vals_reduced[2 * tid];
    xdy_dy_sum_data[2 * out_idx + 1] = vals_reduced[2 * tid + 1];
  }
}

template <typename T>
__global__ void
height_reduce(
    T *xdy_dy_sum_data, // no need to specify T_ACC as T is already an accumulation type
    const int H,
    const int C,
    T *xdy_sum_data,
    T *dy_sum_data
  ) {
  /*
  Same thing as width_reduce but over the H (height) instead of the width dimension.
  grid: (x=N, y=C); block: (x=2H/f)
  X shape: (N, C, H, 2) -view-> (N, C, f, H/f, 2); X stride: (2CH, 2H, 2H/f, H/f, 1)
  dram reduction (per block): (f, H/f, 2) -reduce-> (H/f, 2)
  shmem reduction (per block): (H/f, 2) -reduce-> (2,)
  output buffer: (N, C, 2)
  */
  const int TPB = blockDim.x;
  const int tid = threadIdx.x;

  // shmem reduction
  extern __shared__ char vals_reduced_uncasted[];
  T *vals_reduced = reinterpret_cast<T*>(vals_reduced_uncasted);
  T sum = 0;
  int i;
#pragma unroll
  for (i = 0; i < ceil((float)2 * H / TPB) - 1; ++i) {
    int idx = 0;
    idx += blockIdx.x * C * H * 2;
    idx += blockIdx.y * H * 2;
    idx += i * TPB;
    idx += tid;
    sum += xdy_dy_sum_data[idx];
  }
  if (i * TPB + tid < 2 * H)
    sum += xdy_dy_sum_data[blockIdx.x * C * H * 2 + blockIdx.y * H * 2 + i * TPB + tid];

  vals_reduced[tid] = sum;
  __syncthreads();
  sum_reduce(vals_reduced, TPB / 2, 2);

  // put reduced outputs into return buffers
  if (tid == 0) {
    int out_idx = blockIdx.x * C + blockIdx.y;
    xdy_sum_data[out_idx] = vals_reduced[0];
    dy_sum_data[out_idx] = vals_reduced[1];
  }
}

template <typename T>
__global__ void
compute_bwd_scale_biases(
    const T* mean_data,
    const T* rstd_data,
    const T* weight_data,
    const T* bias_data,
    typename acc_type<T>::type* xdy_sum_data,
    typename acc_type<T>::type* dy_sum_data,
    const int H,
    const int W,
    const int C,
    const int G,
    typename acc_type<T>::type* coef1_data,
    typename acc_type<T>::type* coef2_data,
    typename acc_type<T>::type* coef3_data,
    typename acc_type<T>::type* coef4_data
    ) {
  /*
  Calculates coefficients to reduce computation on the elementwise kernel.
  - coef1: fused scale (rstd * weight)
  - coef2: fused bias (-mean * rstd * weight + bias)
  - coef3/4: some derivative terms
  griddim: (x=N, y=f); blockdim: (x=C/f)
  - d = num. spatial elements (from HW dimension) each thread-block processes in parallel
  - Cd = TPB (threads per block)
  X shape: (N, C) -view-> (N, G, D) -permute-> (N, D, G) -reduce-> (N, G)
  shmem reduction: (D, G) -reduce-> G
  output buffer: (N, G)
  */
  using T_ACC = typename acc_type<T>::type;
  const int D = C / G;
  const int f = gridDim.y;
  const int Gf = G / f;
  const int n = blockIdx.x;
  const int c = blockIdx.y * blockDim.x + threadIdx.x;
  const int g = c / D;
  const int d = c % D;
  const int nc = n * C + c;
  const T_ACC gamma_v = static_cast<T_ACC>(weight_data[c]);

  extern __shared__ char vals_reduced_uncasted[]; // size 2*C, C for sum1, C for sum2
  T_ACC *vals_reduced = reinterpret_cast<T_ACC*>(vals_reduced_uncasted);

  int idx = 0;
  idx += d * G / f;
  idx += g % Gf;
  vals_reduced[2 * idx] = xdy_sum_data[nc] * gamma_v;
  vals_reduced[2 * idx + 1] = dy_sum_data[nc] * gamma_v;
  __syncthreads();
  sum_reduce(vals_reduced, C / f, 2 * G / f);

  const int ng = n * G + g;
  const T_ACC mean_elem = static_cast<T_ACC>(mean_data[ng]);
  const T_ACC rstd_elem = static_cast<T_ACC>(rstd_data[ng]);
  coef1_data[nc] = rstd_elem * weight_data[c];
  coef2_data[nc] = -mean_elem * rstd_elem * weight_data[c] + bias_data[c];

  if (d == 0) {
    const T_ACC sum1 = vals_reduced[2 * (g % Gf)];
    const T_ACC sum2 = vals_reduced[2 * (g % Gf) + 1];
    const T_ACC s = T_ACC(1) / static_cast<T_ACC>(D * H * W);
    const T_ACC x = (sum2 * mean_elem - sum1) * rstd_elem * rstd_elem * rstd_elem * s;
    coef3_data[ng] = x;
    coef4_data[ng] = (-x * mean_elem) - (sum2 * s * rstd_elem);
  }
}

template <typename T>
__global__ void
compute_dweight_dbias(
    const T* mean_data,
    const T* rstd_data,
    typename acc_type<T>::type *xdy_sum_data,
    typename acc_type<T>::type *dy_sum_data,
    const int N,
    const int C,
    const int G,
    T* dweight_data,
    T* dbias_data) {
  /*
  Computes derivatives wrt the weight and bias. 
  grid: (x=f), block: (x=C/f)
  */
  using T_ACC = typename acc_type<T>::type;
  const int c = blockIdx.x * blockDim.x + threadIdx.x;
  const int D = C / G;
  const int g = c / D;
  T_ACC sum1 = 0;
  T_ACC sum2 = 0;

#pragma unroll
  for (int n = 0; n < N; ++n) {
    const int nc = n * C + c;
    const int ng = n * G + g;
    sum1 += (xdy_sum_data[nc] - dy_sum_data[nc] * mean_data[ng]) * rstd_data[ng];
    sum2 += dy_sum_data[nc];
  }
  dweight_data[c] = sum1;
  dbias_data[c] = sum2;
}

template <typename T, int LOOP_I, int vec_elems, int64_t act_fn_option>
__global__ void
dx_elem_kernel(
    const T* dy_data,
    const T* X_data,
    typename acc_type<T>::type* coef1_data,
    typename acc_type<T>::type* coef2_data,
    typename acc_type<T>::type* coef3_data,
    typename acc_type<T>::type* coef4_data,
    const int N,
    const int C,
    const int G,
    T* dx_data
    ) {
  /*
  Performs elementwise kernel to calculate gradients wrt X. Vectorized for speed.
  LOOP_I: number of elements that each thread processes.
  vec_elems: number of elements stored for each vector.
  grid: (x=NHWC / (TPB * LOOP_I * f), y=f), block: (x=TPB)
  - HWC % (TPB * LOOP_I * f) = 0
  - TPB * f % C = 0
  X shape: (N, H, W, C) -view-> (NHWC / (TPB * LOOP_I * f), LOOP_I, f, TPB); X.stride: (LOOP_I * f * TPB, f * TPB, TPB, 1)
  */
  using T_ACC = typename acc_type<T>::type;
  using V = float_vec<T, vec_elems>;
  using V_ACC = float_vec<T_ACC, vec_elems>;
  const int f = gridDim.y;
  const int n = (N * blockIdx.x) / gridDim.x;
  const int c = (blockIdx.y * blockDim.x + threadIdx.x) % (C / vec_elems);
  const int g = (G * c) / (C / vec_elems);
  const int nc = n * (C / vec_elems) + c;
  const int ng = n * G + g;
  T_ACC coef3 = coef3_data[ng];
  T_ACC coef4 = coef4_data[ng];
  const V *dy_vecs = reinterpret_cast<const V*>(dy_data);
  const V *X_vecs = reinterpret_cast<const V*>(X_data);
  V *dx_vecs = reinterpret_cast<V*>(dx_data);
  V_ACC coef1_vec = reinterpret_cast<V_ACC*>(coef1_data)[nc];
  V_ACC coef2_vec = reinterpret_cast<V_ACC*>(coef2_data)[nc];

  T (*act_d_fn)(T);
  if constexpr (act_fn_option == 0)
    act_d_fn = identity_d;
  else if constexpr (act_fn_option == 1)
    act_d_fn = relu_d;
  else if constexpr (act_fn_option == 2)
    act_d_fn = silu_d;
  else if constexpr (act_fn_option == 3)
    act_d_fn = gelu_d;
  else if constexpr (act_fn_option == 4)
    act_d_fn = gelu_tanh_d;

#pragma unroll
  for (int i = 0; i < LOOP_I; ++i) {
    int idx = 0;
    idx += blockIdx.x * LOOP_I * f * blockDim.x;
    idx += i * f * blockDim.x;
    idx += blockIdx.y * blockDim.x;
    idx += threadIdx.x;

    V dy_vec = dy_vecs[idx];
    V X_vec = X_vecs[idx];

    if constexpr (vec_elems == 1) {
      V X_norm = {X_vec.x * coef1_vec.x + coef2_vec.x};
      dx_vecs[idx] = {
        (coef1_vec.x * act_d_fn(X_norm.x) * dy_vec.x)
          + ((coef3 * X_vec.x) + coef4)
      };
    }
    else if constexpr (vec_elems == 2) {
      V X_norm = {
        X_vec.x * coef1_vec.x + coef2_vec.x,
        X_vec.y * coef1_vec.y + coef2_vec.y,
      };
      dx_vecs[idx] = {
        (coef1_vec.x * act_d_fn(X_norm.x) * dy_vec.x)
          + ((coef3 * X_vec.x) + coef4),
        (coef1_vec.y * act_d_fn(X_norm.y) * dy_vec.y)
          + ((coef3 * X_vec.y) + coef4),
      };
    }
    else if constexpr (vec_elems == 4) {
      V X_norm = {
        X_vec.x * coef1_vec.x + coef2_vec.x,
        X_vec.y * coef1_vec.y + coef2_vec.y,
        X_vec.z * coef1_vec.z + coef2_vec.z,
        X_vec.w * coef1_vec.w + coef2_vec.w,
      };
      dx_vecs[idx] = {
        (coef1_vec.x * act_d_fn(X_norm.x) * dy_vec.x)
          + ((coef3 * X_vec.x) + coef4),
        (coef1_vec.y * act_d_fn(X_norm.y) * dy_vec.y)
          + ((coef3 * X_vec.y) + coef4),
        (coef1_vec.z * act_d_fn(X_norm.z) * dy_vec.z)
          + ((coef3 * X_vec.z) + coef4),
        (coef1_vec.w * act_d_fn(X_norm.w) * dy_vec.w)
          + ((coef3 * X_vec.w) + coef4),
      };
    }
  }
}

template <typename T>
void run_gn_bwd_kernels(
      const T *dy_data,
      const T *X_data,
      const T *weight_data,
      const T *bias_data,
      const T *mean_data,
      const T *rstd_data,
      const int N,
      const int H,
      const int W,
      const int C,
      const int G,
      const int64_t act_fn_option,
      T *dx_data,
      T *dweight_data,
      T *dbias_data
  ) {
  using T_ACC = typename acc_type<T>::type;
  hipStream_t hip_stream = at::cuda::getCurrentCUDAStream();
  const int D = C / G;

  T_ACC* xdy_dy_sum_data = (T_ACC*)c10::cuda::CUDACachingAllocator::raw_alloc(sizeof(T_ACC) * N * C * H * 2);

  // sum over W dim
  {
    auto [TPB, d, f] = calc_block_params(W * C, C, G);
    DEBUG("starting width reduce, N: %d, H: %d, W: %d, C: %d, G: %d, TPB: %d, d: %d, f: %d\n", N, H, W, C, G, TPB, d, f);
    if (act_fn_option == 0)
      width_reduce<T, 0><<<dim3(N, H, f), dim3(TPB / d, d), sizeof(T_ACC) * 2 * TPB, hip_stream>>>(
          dy_data, X_data, 
          mean_data, rstd_data,
          weight_data, bias_data,
          H, W, C, G,
          xdy_dy_sum_data);
    else if (act_fn_option == 1)
      width_reduce<T, 1><<<dim3(N, H, f), dim3(TPB / d, d), sizeof(T_ACC) * 2 * TPB, hip_stream>>>(dy_data, X_data, mean_data, rstd_data, weight_data, bias_data, H, W, C, G, xdy_dy_sum_data);
    else if (act_fn_option == 2)
      width_reduce<T, 2><<<dim3(N, H, f), dim3(TPB / d, d), sizeof(T_ACC) * 2 * TPB, hip_stream>>>(dy_data, X_data, mean_data, rstd_data, weight_data, bias_data, H, W, C, G, xdy_dy_sum_data);
    else if (act_fn_option == 3)
      width_reduce<T, 3><<<dim3(N, H, f), dim3(TPB / d, d), sizeof(T_ACC) * 2 * TPB, hip_stream>>>(dy_data, X_data, mean_data, rstd_data, weight_data, bias_data, H, W, C, G, xdy_dy_sum_data);
    else if (act_fn_option == 4)
      width_reduce<T, 4><<<dim3(N, H, f), dim3(TPB / d, d), sizeof(T_ACC) * 2 * TPB, hip_stream>>>(dy_data, X_data, mean_data, rstd_data, weight_data, bias_data, H, W, C, G, xdy_dy_sum_data);
  }

  T_ACC* xdy_sum_data = (T_ACC*)c10::cuda::CUDACachingAllocator::raw_alloc(sizeof(T_ACC) * N * C);
  T_ACC* dy_sum_data = (T_ACC*)c10::cuda::CUDACachingAllocator::raw_alloc(sizeof(T_ACC) * N * C);
  // sum over H dim
  {
    auto [TPB, d, f] = calc_block_params(2 * H, 2);
    DEBUG("starting height reduce, N: %d, H: %d, W: %d, C: %d, G: %d, TPB: %d, d: %d, f: %d\n", N, H, W, C, G, TPB, d, f);
    height_reduce<<<dim3(N, C), TPB, sizeof(T_ACC) * TPB, hip_stream>>>(
        xdy_dy_sum_data,
        H, C,
        xdy_sum_data, dy_sum_data);
  }
  c10::cuda::CUDACachingAllocator::raw_delete(xdy_dy_sum_data);

  // compute weight/bias grads
  {
    auto [TPB, d, f] = calc_block_params(C, C, G);
    DEBUG("starting compute dweight dbias, N: %d, H: %d, W: %d, C: %d, G: %d, TPB: %d, d: %d, f: %d\n", N, H, W, C, G, TPB, d, f);
    compute_dweight_dbias<<<f, C / f, 0, hip_stream>>>(
        mean_data, rstd_data,
        xdy_sum_data, dy_sum_data,
        N, C, G,
        dweight_data, dbias_data);
  }

  T_ACC *coef1_data = (T_ACC*)c10::cuda::CUDACachingAllocator::raw_alloc(sizeof(T_ACC) * N * C);
  T_ACC *coef2_data = (T_ACC*)c10::cuda::CUDACachingAllocator::raw_alloc(sizeof(T_ACC) * N * C);
  T_ACC *coef3_data = (T_ACC*)c10::cuda::CUDACachingAllocator::raw_alloc(sizeof(T_ACC) * N * G);
  T_ACC *coef4_data = (T_ACC*)c10::cuda::CUDACachingAllocator::raw_alloc(sizeof(T_ACC) * N * G);
  // compute fused scales/biases for dx elementwise kernel
  {
    auto [TPB, d, f] = calc_block_params(C, C, G);
    DEBUG("starting bwd scale biases, N: %d, H: %d, W: %d, C: %d, G: %d, TPB: %d, d: %d, f: %d\n", N, H, W, C, G, TPB, d, f);
    compute_bwd_scale_biases<<<dim3(N, f), C / f, sizeof(T_ACC) * 2 * C / f, hip_stream>>>(
        mean_data, rstd_data, weight_data, bias_data,
        xdy_sum_data, dy_sum_data,
        H, W, C, G,
        coef1_data, coef2_data, coef3_data, coef4_data);
  }

  {
    int vec_elems;
    if (D % 4 == 0) vec_elems = 4;
    else if (D % 2 == 0) vec_elems = 2;
    else vec_elems = 1;
    auto [TPB, d, f] = calc_block_params(H * W * C, C, G);

    if (!ELEM_DEBUG && ((H * W * C) % (TPB * 8 * f * vec_elems) == 0)) {
      const int LOOP_I = 8;
      const int num_blocks = ceil((float)N * H * W * C / TPB / LOOP_I / f);
      DEBUG("dx elem kernel starting, N: %d, H: %d, W: %d, C: %d, G: %d, D: %d, TPB: %d, f: %d, num blocks (before vectors): %d, vec_elems: %d\n", N, H, W, C, G, D, TPB, f, num_blocks, vec_elems);
      if (D % 4 == 0 && act_fn_option == 0)
        dx_elem_kernel<T, LOOP_I, 4, 0><<<dim3(num_blocks / 4, f), TPB, 0, hip_stream>>>(dy_data, X_data, coef1_data, coef2_data, coef3_data, coef4_data, N,  C, G, dx_data);
      else if (D % 2 == 0 && act_fn_option == 0)
        dx_elem_kernel<T, LOOP_I, 2, 0><<<dim3(num_blocks / 2, f), TPB, 0, hip_stream>>>(dy_data, X_data, coef1_data, coef2_data, coef3_data, coef4_data, N,  C, G, dx_data);
      else if (D % 1 == 0 && act_fn_option == 0)
        dx_elem_kernel<T, LOOP_I, 1, 0><<<dim3(num_blocks / 1, f), TPB, 0, hip_stream>>>(dy_data, X_data, coef1_data, coef2_data, coef3_data, coef4_data, N,  C, G, dx_data);
      else if (D % 4 == 0 && act_fn_option == 1)
        dx_elem_kernel<T, LOOP_I, 4, 1><<<dim3(num_blocks / 4, f), TPB, 0, hip_stream>>>(dy_data, X_data, coef1_data, coef2_data, coef3_data, coef4_data, N,  C, G, dx_data);
      else if (D % 2 == 0 && act_fn_option == 1)
        dx_elem_kernel<T, LOOP_I, 2, 1><<<dim3(num_blocks / 2, f), TPB, 0, hip_stream>>>(dy_data, X_data, coef1_data, coef2_data, coef3_data, coef4_data, N,  C, G, dx_data);
      else if (D % 1 == 0 && act_fn_option == 1)
        dx_elem_kernel<T, LOOP_I, 1, 1><<<dim3(num_blocks / 1, f), TPB, 0, hip_stream>>>(dy_data, X_data, coef1_data, coef2_data, coef3_data, coef4_data, N,  C, G, dx_data);
      else if (D % 4 == 0 && act_fn_option == 2)
        dx_elem_kernel<T, LOOP_I, 4, 2><<<dim3(num_blocks / 4, f), TPB, 0, hip_stream>>>(dy_data, X_data, coef1_data, coef2_data, coef3_data, coef4_data, N,  C, G, dx_data);
      else if (D % 2 == 0 && act_fn_option == 2)
        dx_elem_kernel<T, LOOP_I, 2, 2><<<dim3(num_blocks / 2, f), TPB, 0, hip_stream>>>(dy_data, X_data, coef1_data, coef2_data, coef3_data, coef4_data, N,  C, G, dx_data);
      else if (D % 1 == 0 && act_fn_option == 2)
        dx_elem_kernel<T, LOOP_I, 1, 2><<<dim3(num_blocks / 1, f), TPB, 0, hip_stream>>>(dy_data, X_data, coef1_data, coef2_data, coef3_data, coef4_data, N,  C, G, dx_data);
      else if (D % 4 == 0 && act_fn_option == 3)
        dx_elem_kernel<T, LOOP_I, 4, 3><<<dim3(num_blocks / 4, f), TPB, 0, hip_stream>>>(dy_data, X_data, coef1_data, coef2_data, coef3_data, coef4_data, N,  C, G, dx_data);
      else if (D % 2 == 0 && act_fn_option == 3)
        dx_elem_kernel<T, LOOP_I, 2, 3><<<dim3(num_blocks / 2, f), TPB, 0, hip_stream>>>(dy_data, X_data, coef1_data, coef2_data, coef3_data, coef4_data, N,  C, G, dx_data);
      else if (D % 1 == 0 && act_fn_option == 3)
        dx_elem_kernel<T, LOOP_I, 1, 3><<<dim3(num_blocks / 1, f), TPB, 0, hip_stream>>>(dy_data, X_data, coef1_data, coef2_data, coef3_data, coef4_data, N,  C, G, dx_data);
      else if (D % 4 == 0 && act_fn_option == 4)
        dx_elem_kernel<T, LOOP_I, 4, 4><<<dim3(num_blocks / 4, f), TPB, 0, hip_stream>>>(dy_data, X_data, coef1_data, coef2_data, coef3_data, coef4_data, N,  C, G, dx_data);
      else if (D % 2 == 0 && act_fn_option == 4)
        dx_elem_kernel<T, LOOP_I, 2, 4><<<dim3(num_blocks / 2, f), TPB, 0, hip_stream>>>(dy_data, X_data, coef1_data, coef2_data, coef3_data, coef4_data, N,  C, G, dx_data);
      else if (D % 1 == 0 && act_fn_option == 4)
        dx_elem_kernel<T, LOOP_I, 1, 4><<<dim3(num_blocks / 1, f), TPB, 0, hip_stream>>>(dy_data, X_data, coef1_data, coef2_data, coef3_data, coef4_data, N,  C, G, dx_data);
    }
    else { // relatively slow fallback
      const int num_blocks = N * H * W;
      DEBUG("SLOW FALLBACK, dx elem kernel starting, N: %d, H: %d, W: %d, C: %d, G: %d, D: %d, TPB: %d, f: %d, num blocks (before vectors): %d, vec_elems: %d\n", N, H, W, C, G, D, C/f, f, num_blocks, vec_elems);
      if (act_fn_option == 0)
        dx_elem_kernel<T, 1, 1, 0><<<dim3(num_blocks, f), C / f, 0, hip_stream>>>(dy_data, X_data, coef1_data, coef2_data, coef3_data, coef4_data, N,  C, G, dx_data);
      else if (act_fn_option == 1)
        dx_elem_kernel<T, 1, 1, 1><<<dim3(num_blocks, f), C / f, 0, hip_stream>>>(dy_data, X_data, coef1_data, coef2_data, coef3_data, coef4_data, N,  C, G, dx_data);
      else if (act_fn_option == 2)
        dx_elem_kernel<T, 1, 1, 2><<<dim3(num_blocks, f), C / f, 0, hip_stream>>>(dy_data, X_data, coef1_data, coef2_data, coef3_data, coef4_data, N,  C, G, dx_data);
      else if (act_fn_option == 3)
        dx_elem_kernel<T, 1, 1, 3><<<dim3(num_blocks, f), C / f, 0, hip_stream>>>(dy_data, X_data, coef1_data, coef2_data, coef3_data, coef4_data, N,  C, G, dx_data);
      else if (act_fn_option == 4)
        dx_elem_kernel<T, 1, 1, 4><<<dim3(num_blocks, f), C / f, 0, hip_stream>>>(dy_data, X_data, coef1_data, coef2_data, coef3_data, coef4_data, N,  C, G, dx_data);
    }
  }

  c10::cuda::CUDACachingAllocator::raw_delete(xdy_sum_data);
  c10::cuda::CUDACachingAllocator::raw_delete(dy_sum_data);
  c10::cuda::CUDACachingAllocator::raw_delete(coef1_data);
  c10::cuda::CUDACachingAllocator::raw_delete(coef2_data);
  c10::cuda::CUDACachingAllocator::raw_delete(coef3_data);
  c10::cuda::CUDACachingAllocator::raw_delete(coef4_data);
}

template void run_gn_bwd_kernels<double>(const double *dy_data, const double *X_data, const double *weight_data, const double *bias_data, const double *mean_data, const double *rstd_data, const int N, const int H, const int W, const int C, const int G, const int64_t act_fn_option, double *dx_data, double *dweight_data, double *dbias_data);
template void run_gn_bwd_kernels<float>(const float *dy_data, const float *X_data, const float *weight_data, const float *bias_data, const float *mean_data, const float *rstd_data, const int N, const int H, const int W, const int C, const int G, const int64_t act_fn_option, float *dx_data, float *dweight_data, float *dbias_data);
template void run_gn_bwd_kernels<c10::Half>(const c10::Half *dy_data, const c10::Half *X_data, const c10::Half *weight_data, const c10::Half *bias_data, const c10::Half *mean_data, const c10::Half *rstd_data, const int N, const int H, const int W, const int C, const int G, const int64_t act_fn_option, c10::Half *dx_data, c10::Half *dweight_data, c10::Half *dbias_data);
template void run_gn_bwd_kernels<c10::BFloat16>(const c10::BFloat16 *dy_data, const c10::BFloat16 *X_data, const c10::BFloat16 *weight_data, const c10::BFloat16 *bias_data, const c10::BFloat16 *mean_data, const c10::BFloat16 *rstd_data, const int N, const int H, const int W, const int C, const int G, const int64_t act_fn_option, c10::BFloat16 *dx_data, c10::BFloat16 *dweight_data, c10::BFloat16 *dbias_data);
